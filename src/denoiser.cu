#include "hip/hip_runtime.h"
#include <denoiser.hpp>
#include <iomanip>

// -- Inline function to get all octancts in a vector
__host__
inline void get_octancts(vector<unsigned char>& vec, const unsigned char& pos_flag, const int& p)
{
    // Flag that implies that p < 0
    const unsigned char neg_flag = 0b000;

    // Calculate the corresponding octanct
    const unsigned char oct = (p > 0) ? pos_flag : neg_flag;

    // Append the value into the vector
    vec.push_back(oct);

    // If p is zero, then append the masked octanct as well
    if (p == 0) vec.push_back(oct ^ pos_flag);
}

// -- Indices whose distance to a central point is less than R_max {{{
__host__
vector<grid_point> Denoiser::table_of_indices(Map& map, const float& r_env) 
{
    // Vector that will conatin all nearest indices
    vector<grid_point> indices;

    map.grid.use_points_around(Fractional(0, 0, 0), r_env,
        [&map, &indices](const float& m, const double& d2, const Position& dp)
        {
            indices.push_back(map.grid.get_nearest_point(dp));
        }
    );

    return indices;
}
// -- }}}

// -- Construct the environment around a given grid point
__host__
vector<float>* Denoiser::get_octs(
    const Map& map, const int& u, const int& v, const int& w, 
    const vector<grid_point>& indices
) {
    // Vector containing the points per octanct
    vector<float>* oct_points = new vector<float>[Octanct::No];

    // Vectors used to assign points to octancts
    vector<octanct> u_oct, v_oct, w_oct;

    // Constants to avoid magic numbers
    const unsigned char positive_u_flag = 0b001; // 1
    const unsigned char positive_v_flag = 0b010; // 2
    const unsigned char positive_w_flag = 0b100; // 4

    // Iterate for all points in the table
    for (auto& p : indices) {

        // Obtain the right octancts in the u direction
        get_octancts(u_oct, positive_u_flag, p.u);

        // Obtain the right octancts in the v direction
        get_octancts(v_oct, positive_v_flag, p.v);

        // Obtain the right octancts in the w direction
        get_octancts(w_oct, positive_w_flag, p.w);

        // Normalisation factor used to divide points among octancts
        const int norm = u_oct.size() * v_oct.size() * w_oct.size();

        // Append the normalised map value to the relevant octancts
        for (auto& ou : u_oct) {
            for (auto& ov : v_oct) {
                for (auto& ow : w_oct) {
                    oct_points[(ow | ov | ou)].push_back(
                        map.get_value(p.u + u, p.v + v, p.w + w) / norm
                    );
                }
            }
        }

        // Clear the memory in all octanct vectors for next iteration
        u_oct.clear(); v_oct.clear(); w_oct.clear();
    }

    return oct_points;
}
// -- }}}

// -- Get the average points per octanct {{{
__host__
int Denoiser::avg_points_per_octanct(Map& map, const float& r_env)
{
    // Obtain the table of indices
    const auto indices = table_of_indices(map, r_env);

    // Vectors that will contain the points
    vector<int> oct_points[Octanct::No];

    // Vector that will contain the average of each octanct
    vector<float> pp_oct(Octanct::No);

    // Vectors used to assign points to quadrants
    vector<octanct> u_oct, v_oct, w_oct;

    // Constants to avoid magic numbers
    const unsigned char positive_u_flag = 0b001; // 1
    const unsigned char positive_v_flag = 0b010; // 2
    const unsigned char positive_w_flag = 0b100; // 4

    // Iterate for all points in the table
    for (auto& p : indices) {

        // Obtain the right octancts in the u direction
        get_octancts(u_oct, positive_u_flag, p.u);

        // Obtain the right octancts in the v direction
        get_octancts(v_oct, positive_v_flag, p.v);

        // Obtain the right octancts in the w direction
        get_octancts(w_oct, positive_w_flag, p.w);

        // Add one count to each relevant quadrant
        for (auto& ou : u_oct) {
            for (auto& ov : v_oct) {
                for (auto& ow : w_oct) {
                    oct_points[(ow | ov | ou)].push_back(1);
                }
            }
        }

        // Clear the vectors for the next iteration
        u_oct.clear(); v_oct.clear(); w_oct.clear();
    }

    // Calculate the average number of points per octact
    float avg_count = 0.0f;

    // Get the number of points per octanct
    for (octanct o = 0; o < Octanct::No; o++) {
        avg_count += oct_points[o].size();
    }

    return avg_count / Octanct::No;
}
// -- }}}

// -- Table containing the environment data, its average and standard deviation {{{
__host__
float* Denoiser::table_of_envs(Map& map, const float& r_env)
{
    // First, obtain a table of near indices
    const auto indices = table_of_indices(map, r_env);

    // Number of rows and columns in the array
    const int& Ne = map.get_volume();
    const int& No = Octanct::No;

    // Allocate memory for all octancts in the grid
    float* envs = new float[Ne * No];

    // Count the corresponding environment for each point
    int eidx = 0;

    // iterate for each point in the grid to obtain its environment
    for (int w = 0; w < map.Nw; w++) {
        for (int v = 0; v < map.Nv; v++) {
            for (int u = 0; u < map.Nu; u++) {

                // Get all points in each octanct
                const vector<float>* oct_points = get_octs(map, u, v, w, indices);

                // Vector containing the sum of all points in each octanct
                float oct_sum[Octanct::No];

                // Iterate for each octanct to calculate its average value
                for (int o = 0; o < Octanct::No; o++) {

                    // Sum of all points in the current octanct
                    oct_sum[o] = std::accumulate(
                        oct_points[o].begin(), oct_points[o].end(), 0.0f
                    );

                    // Copy the octanct average to the correct environment
                    envs[eidx * No + o] = oct_sum[o] / oct_points[o].size();
                }

                // Move to the next environment in the grid
                eidx++;

                // Delete the heap allocated data
                delete[] oct_points;
            }
        }
    }

    // Return the table of environments
    return envs;
}
// -- }}}

// -- Table containing environment averages {{{
__host__
vector<float> Denoiser::table_of_stats(Map& map, const float& r_env)
{
    // First, obtain a table of near indices
    const auto indices = table_of_indices(map, r_env);

    // Number of rows and columns in the array
    const int Ne = map.get_volume();

    // Allocate memory for all octancts in the grid
    vector<float> env_stats(Ne);

    // Count the corresponding environment for each point
    int eidx = 0;

    // iterate for each point in the grid to obtain its environment
    for (int w = 0; w < map.Nw; w++) {
        for (int v = 0; v < map.Nv; v++) {
            for (int u = 0; u < map.Nu; u++) {

                // Get all points in each octanct
                const vector<float>* oct_points = get_octs(map, u, v, w, indices);

                // Vector containing the sum of all points in each octanct
                float oct_sum[Octanct::No];

                // Temporary that will contain the avg of the environment
                float env_avg = 0.0f;

                // Iterate for each octanct to calculate its average value
                for (int o = 0; o < Octanct::No; o++) {

                    // Sum of all points in the current octanct
                    oct_sum[o] = std::accumulate(
                        oct_points[o].begin(), oct_points[o].end(), 0.0f
                    );

                    // Add the value to the environment average
                    env_avg += oct_sum[o];
                }

                // Calculate the average of the environment
                env_stats[eidx] = env_avg / indices.size();

                // Move to the next environment in the grid
                eidx++;

                // Delete the heap allocated data
                delete[] oct_points;
            }
        }
    }

    // Return the table of environments
    return env_stats;
}

__host__
vector<float> Denoiser::table_of_stats(Map& map, const float* envs, const float& r_env)
{
    // First, obtain a table of near indices
    const auto indices = table_of_indices(map, r_env);

    // Number of rows and columns in the array
    const int Ne = map.get_volume();

    // Allocate memory for all octancts in the grid
    vector<float> env_stats(Ne);

    // Count the corresponding environment for each point
    int eidx = 0;

    // iterate for each point in the grid to obtain its environment
    for (int w = 0; w < map.Nw; w++) {
        for (int v = 0; v < map.Nv; v++) {
            for (int u = 0; u < map.Nu; u++) {

                // Temporary that will contain the avg of the environment
                float env_avg = 0.0f;

                // Add each octanct to the environment average
                for (int o = 0; o < Octanct::No; o++) {
                    env_avg += envs[eidx * Octanct::No + o];
                }

                // Calculate the average of the environment
                env_stats[eidx] = env_avg / indices.size();

                // Move to the next environment in the grid
                eidx++;
            }
        }
    }

    // Return the table of environments
    return env_stats;
}
// -- }}}

// -- Main algorithm to denoise a map using non-local means {{{
__host__
std::tuple<Map, float> Denoiser::nlmeans_denoiser(
    Map& map, const float& p_thresh, const float& r_env
) {
    // Construct some needed aliases
    const int& Ne = map.get_volume(); // -- Number of environments (points) in the map
    const int& No = Octanct::No;      // -- Number of octancts in an env (8)
    const int& Nr = Octanct::Nr;      // -- Number of rotations per comp (10)

    // Generate a copy of the map to denoise it
    Map denoised_map = map;

    // Pointers to the denoised map and original map memory blocks
    float* denoised_M = denoised_map.data();
    float* original_M = map.data();

    // Block of memory containing all environments and their averages
    const float* envs = table_of_envs(map, r_env);

    // Table containing the rotated indices for each needed rotation
    const octanct* rots = Octanct::table_of_rotations();

    // Get all average environments of the map using envs -- std::vector
    const auto env_avg   = table_of_stats(map, envs, r_env);

    // Get the maximum and minimum environment average
    const auto min = std::min_element(env_avg.begin(), env_avg.end());
    const auto max = std::max_element(env_avg.begin(), env_avg.end());

    // Calculate the denoising parameter using the threshold provided
    const float hd      = 0.5 * p_thresh * (*max - *min);
    const float inv_den = 1 / (2 * hd * hd);

    // Generate the device copies of the relevant objects
    float* d_omap; float* d_dmap; float* d_envs; 
    octanct* d_rots; float* d_sumk; float* d_dsq;

    // Allocate some memory for the needed objects
    hipMalloc(&d_omap, Ne * sizeof(float));        // -- Original map
    hipMalloc(&d_dmap, Ne * sizeof(float));        // -- Denoised map
    hipMalloc(&d_sumk, Ne * sizeof(float));        // -- Sum of kernels
    hipMalloc(&d_envs, Ne * No * sizeof(float));   // -- Environments in the map
    hipMalloc(&d_rots, Nr * No * sizeof(octanct)); // -- Table of rotations
    hipMalloc(&d_dsq,  Ne * Nr * sizeof(float));   // -- Distance squared values

    // Copy the original map, the environments and the table of rotations
    hipMemcpy(d_envs, envs,       Ne * No * sizeof(float),   hipMemcpyHostToDevice);
    hipMemcpy(d_omap, original_M, Ne * sizeof(float),        hipMemcpyHostToDevice);
    hipMemcpy(d_rots, rots,       Nr * No * sizeof(octanct), hipMemcpyHostToDevice);

    // Set the denoised map and the sum of kernels to zero
    hipMemset(d_dmap, 0.0f, Ne * sizeof(float));
    hipMemset(d_sumk, 0.0f, Ne * sizeof(float));

    // Iterate through all reference environments in the map
    for (int er = 0; er < Ne; er++) {

        // Generate the geometry of the blocks to compute the distance squared
        int T_dsq = 160;
        int B_dsq = (Ne - er) / 2 + 1;

        // Generate the geometry of the blocks to update the denoiser
        int T_den = 128;
        int B_den = (Ne - er) / T_den + 1;

        // Set enough memory in d_dsq to zero to compute the new distances
        hipMemset(d_dsq, 0.0f, (Ne - er) * Nr * sizeof(float));

        // Calculate all possible distance squared in parallel
        calculate_dsq<<<B_dsq, T_dsq>>>(d_dsq, d_envs, d_rots, er, Ne, Nr, No);

        // Update the denoised map and the sum of kernels
        update_denoiser<<<B_den, T_den>>>(
            d_dmap, d_sumk, d_dsq, d_omap, er, Ne, Nr, inv_den
        );
    } // -- End of the denoiser loop

    // Host allocated version of the sum of kernels
    float* sum_kernels = new float[Ne];

    // Copy the sum of kernels and the denoised map to the host
    hipMemcpy(denoised_M,  d_dmap, Ne * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(sum_kernels, d_sumk, Ne * sizeof(float), hipMemcpyDeviceToHost);

    // Normalise the data using the sum of kernels
    for (int er = 0; er < Ne; er++) {
        denoised_M[er] = denoised_M[er] / sum_kernels[er];
    }

    // Delete the heap allocated data
    delete[] envs;
    delete[] rots;
    delete[] sum_kernels;

    // Delete the device allocated data
    hipFree(d_omap);
    hipFree(d_dmap);
    hipFree(d_envs);
    hipFree(d_rots);
    hipFree(d_sumk);

    // Return a tuple containing the denoised map and the denoised parameter
    return std::make_tuple(denoised_map, hd);
}
// -- }}}

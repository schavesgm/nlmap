#include "hip/hip_runtime.h"
#include <cudenoiser.hpp>

__forceinline__ __device__
int __get_oct(const int& t, const int& No)
{
    // -- Obtain the corresponding octanct for a given thread using the 2^n 
    // -- optimised modulo operator. The calculation is equivalent as t % No
    return t & (No - 1);
}

__forceinline__ __device__
int __get_rot(const int& t, const int& Nr, const int& No)
{
    // -- Obtain the corresponding rotation for a given thread using. Each block
    // -- contains 160 threads, so there are 2 comparisons of 10 rotations each
    return (t >= Nr * No) ? (t - Nr * No) / No : t / No;
}

__forceinline__ __device__
int __get_idc(const int& b, const int& t, const int& Nr, const int& No)
{
    // -- Obtain the corresponding index of comparison for the given thread and 
    // -- the given block. For each block, there are 2 comparisons.
    return b * 2 + (t >= Nr * No);
}

__global__
void Cudenoiser::calculate_dsq(
    float* dsq, float* envs, const octanct* rots, const int er, 
    const int Ne, const int Nr, const int No
) {
    // -- Calculate all distance squared for all possible rotations using er as
    // -- the reference environment and ec = [er, Ne] as the comparison environments.
    // -- Each thread in the calculation will performn one comparison for one octact,
    // -- rotation and comparison environment. It will be stored in the block of
    // --  memory dsq with dimensions (Ne - er, Nr)

    // Obtain the global index of the current thread
    const int gidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Only use the threads that are inside bounds (Ne - er) * Nr * No
    if (gidx < (Ne - er) * Nr * No) {

        // Calculate the corresponding octact for the current thread
        const int oct = __get_oct(threadIdx.x, No);

        // Calculate the corresponding rotation for the current thread
        const int rot = __get_rot(threadIdx.x, Nr, No);

        // Calculate the comparison index for the current thread
        const int idc = __get_idc(blockIdx.x, threadIdx.x, Nr, No);

        // Compute the comparison environment ec = er + idc
        const int ec = er + idc;

        // Get a reference to the rotated reference octanct and comparison
        const float& r_oct = envs[er * No + rots[rot * No + oct]];
        const float& c_oct = envs[ec * No + oct];

        // Compute the octanct different between reference and comparison
        const float diff = r_oct - c_oct;

        // Atomic add the difference to the correct location
        atomicAdd(dsq + idc * Nr + rot, (diff * diff) / No);
    }
}

__global__
void Cudenoiser::update_denoiser(
    float* dmap, float* kernels, float* d_squared, float* omap,
    const int er, const int Ne, const int Nr, const float inv_den
) {
    // -- Update the denoised map and the sum of kernels using uhat and kernel 
    // -- at both relevant positions (er and ec). The kernel is computed by first
    // -- finding the minimum distance squared in the matrix of dimensions 
    // -- (Ne - er, Nr) for each row. Each thread in the calculation will find the
    // -- minimum value of each different row and then compute the kernel and u_hat,
    // --  kernel = exp(-min_dsq * inv_den), u_hat[e] = kernel[e] * map_values[e]

    // Get the global index of the current thread -- Corresponds to idc (previously)
    const int gidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the current thread is inside the bounds
    if (gidx < (Ne - er)) {

        // Generate a temporary variable to find the minimum distance squared
        float min_dsq = d_squared[gidx * Nr + 0];

        // Update the minimum value if needed
        for (int r = 1; r < Nr; r++) {
            if (min_dsq > d_squared[gidx * Nr + r]) {
                min_dsq = d_squared[gidx * Nr + r];
            }
        }

        // Compute the kernel using the minimum distance
        float kernel = expf(- min_dsq * inv_den);

        // Update the denoised map in the correct locations
        atomicAdd(dmap + gidx + er, kernel * omap[er]);
        atomicAdd(dmap + er,        kernel * omap[gidx + er]);

        // Add the kernels to the correct locations
        atomicAdd(kernels + gidx + er, kernel);
        atomicAdd(kernels + er,        kernel);
    }
}

#include "hip/hip_runtime.h"
#include <iostream>
#include <tuple>

// User defined modules
#include <Map.hpp>
#include <Argparser.hpp>
#include <path.hpp>
#include <denoiser.hpp>
#include <utils.hpp>
#include <stats.hpp>

int main(const int argc, char** argv)
{
    // Generate an argparser object to deal with command line input
    const Argparser command_args(argc, argv);

    // Check if the argparser contains the --help flag
    const bool is_help = command_args.check_flag("--help");

    if (is_help) {
        std::cout << 
        "  -- denoise_map\n"
        "  Usage:\n"
        "  denoise map --path [str] --name [str] --s [float] --p [float] --r [float]\n\n"
        "  Arguments:\n"
        "   --path: Path where the data is stored.   Example: ./data/rnase\n"
        "   --name: Name of the map file to process. Example: ./refmac.map\n"
        "   --s:    Standard deviation of the noise add. If zero, no noise added\n"
        "   --p:    Percentage of the total spread of the map used to create the\n"
        "           denoiser parameter.\n"
        "   --r:    Radious of search used to create an environment.\n"
        "  Example:\n"
        "  denoise_map --path data/rnase --name refmac.map --s 0.0 --p 0.05 --r 2.0\n\n";
        return 0;
    }

    // Check the existence of several flags in the system
    const bool is_path  = command_args.check_flag("--path");
    const bool is_name  = command_args.check_flag("--name");
    const bool is_s     = command_args.check_flag("--s");
    const bool is_p     = command_args.check_flag("--p");
    const bool is_r     = command_args.check_flag("--r");

    if (!is_path || !is_name || !is_s || !is_p || !is_r) {
        std::cout << " ERROR: Command line arguments are incorrect\n";
        return 1;
    }

    // Get the correct data from the argument parser
    const std::string protein_path = command_args.get_flag("--path");
    const std::string map_name     = command_args.get_flag("--name");
    const float sigma              = command_args.get_flag<float>("--s");
    const float perc_t             = command_args.get_flag<float>("--p");
    const float r_env              = command_args.get_flag<float>("--r");

    // Obtain the name of the protein from the protein path
    const auto protein = Path::get_basename(protein_path);

    // Load a Map file from memory
    Map original_map(Path::join_path(protein_path, map_name));

    // Add some noise to the map according to sigma
    original_map.add_noise(sigma);

    // Denoise the map using the map denoiser
    auto denoiser_output = Denoiser::nlmeans_denoiser(original_map, perc_t, r_env);

    // References to the objects encoded in the denoiser output
    auto& denoised_map       = std::get<0>(denoiser_output);
    auto& denoise_param      = std::get<1>(denoiser_output);

    // Calculate the environment statistics of the noisy and denoised maps
    auto noisy_env_stats    = Denoiser::table_of_stats(original_map, r_env);
    auto denoised_env_stats = Denoiser::table_of_stats(denoised_map, r_env);

    // Generate the path where the maps will be stored
    const auto maps_path = Path::format_str(
        "out/data/%s/s%.4f_h%.4f_r%.4f_p%.4f",
        protein.c_str(), sigma, denoise_param, r_env, perc_t
    );

    // Generate the path where the log will be output
    const auto logs_path = Path::format_str(
       "out/log/%s/s%.4f_h%.4f_r%.4f_p%.4f",
       protein.c_str(), sigma, denoise_param, r_env, perc_t
    );

    // Create the basic directories if needed
    Path::make_path(maps_path); 
    Path::make_path(logs_path);

    // Paths to the noisy and denoised data
    const auto n_files_path = Path::join_path(maps_path, "noisy/files");
    const auto d_files_path = Path::join_path(maps_path, "denoised/files");

    // Paths to the noisy and denoised logs
    const auto n_log_path = Path::join_path(maps_path, "noisy/log");
    const auto d_log_path = Path::join_path(maps_path, "denoised/log");

    // Create the needed directories
    Path::make_path(n_files_path); Path::make_path(n_log_path);
    Path::make_path(d_files_path); Path::make_path(d_log_path);

    // Save the noisy and denoised maps in memory
    original_map.save_map(Path::join_path(n_files_path, "noisy.map"));
    denoised_map.save_map(Path::join_path(d_files_path, "denoised.map"));

    // Save the statistics of the environment in memory
    Utils::save_envstats(
        Path::join_path(n_log_path, "envstats.dat"), 
        noisy_env_stats, original_map
    );

    // Save the average for each environment in the denoised map
    Utils::save_envstats(
        Path::join_path(d_log_path, "envstats.dat"), 
        denoised_env_stats, denoised_map
    );

    // Output the value of h to capture it in the pipeline
    std::cout << denoise_param << std::endl;

return 0;
}
